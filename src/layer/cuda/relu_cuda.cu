#include "hip/hip_runtime.h"
#include "relu_cuda.h"
#include <hip/hip_runtime.h>

namespace ncnn {
    /**
     * CUDA�汾�����kernel() �� CUDA�汾�����kernel��CPP API�ӿ�
     * @param input_blob ��������-��������ָ��
     * @param output_blob �������-��������ָ��
     * @param number ��������Ԫ�ظ���
     * ��CUDA kernel�x(Ҳ���� const float* x)ֻ��һ��ָ�������ڴ����������ָ�룬������֪��Ҳ������ά�ȣ�shape����
     */
    __global__ void relu_kernel_cuda(const float* input_blob, float* output_blob, int number)
    {
        // ����ȫ���߳�������global thread index��
        unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x * 4;
        unsigned int idxElement = idx * 4;
        if (idxElement + 3 < number) {
            float4 vec_in = *(float4*)&input_blob[idxElement];
            float4 vec_out;

            vec_out.x = vec_in.x > 0.0f ? vec_in.x : 0.0f;
            vec_out.y = vec_in.y > 0.0f ? vec_in.y : 0.0f;
            vec_out.z = vec_in.z > 0.0f ? vec_in.z : 0.0f;
            vec_out.w = vec_in.w > 0.0f ? vec_in.w : 0.0f;

            *(float4*)&output_blob[idxElement] = vec_out;
        }
        else {
            for (int i = 0; i < 4; i++) {
                unsigned int elem_idx = idxElement + i;
                if (elem_idx < number) {
                    output_blob[elem_idx] = input_blob[elem_idx] > 0.0f ? input_blob[elem_idx] : 0.0f;
                }
            }
        }
    }
   void relu_cuda(const float* input_blob, float* output_blob, int number)
    {
        // ����ÿ���߳̿���߳�����
        int threadsPerBlock = 1024;
        // �����ܹ���Ҫ���ٸ��߳��������������顣��Ϊÿ���̴߳���4��Ԫ�أ��������߳���Ӧ����Ԫ����������4��������ȡ��������Ϊfloat��4�ֽڣ�16�ֽ�/4�ֽ�=4��
        int totalThreadsNeeded = (number + 4 - 1) / 4;
        // �����������߳̿��������ÿ���߳̿���256���̣߳��������߳�������ÿ���߳̿���߳�����������ȡ����
        int blocksPerGrid = (totalThreadsNeeded + threadsPerBlock - 1) / threadsPerBlock;
        relu_kernel_cuda<<<blocksPerGrid, threadsPerBlock>>>(input_blob, output_blob, number);
        // ͬ���豸���ȴ��ں�ִ����ɡ��������������������CPU��ֱ���豸��GPU���ϵ����в�����ɡ�
        hipDeviceSynchronize();
    }
}