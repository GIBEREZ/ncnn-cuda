#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "relu_cuda.h"

/**
 * CUDA�汾�����kernel() �� CUDA�汾�����kernel��CPP API�ӿ�
 * @param input_blob ��������-��������ָ��
 * @param output_blob �������-��������ָ��
 * @param number ��������Ԫ�ظ���
 * ��CUDA kernel�x(Ҳ���� const float* x)ֻ��һ��ָ�������ڴ����������ָ�룬������֪��Ҳ������ά�ȣ�shape����
 */
__global__ void relu_kernel(const float* input_blob, float* output_blob, int number)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < number) output_blob[idx] = input_blob[idx] > 0.0f ? input_blob[idx] : 0.0f;
}
