#include "hip/hip_runtime.h"
#include "relu_cuda.h"

/**
 * CUDA�汾�����kernel() �� CUDA�汾�����kernel��CPP API�ӿ�
 * @param input_blob ��������-��������ָ��
 * @param output_blob �������-��������ָ��
 * @param number ��������Ԫ�ظ���
 * ��CUDA kernel�x(Ҳ���� const float* x)ֻ��һ��ָ�������ڴ����������ָ�룬������֪��Ҳ������ά�ȣ�shape����
 */
__global__ void relu_kernel_cuda(const float* input_blob, float* output_blob, int number)
{
    // ����ȫ���߳�������global thread index��
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < number) output_blob[idx] = input_blob[idx] > 0.0f ? input_blob[idx] : 0.0f;
}

extern "C" void relu_cuda(const float* input_blob, float* output_blob, int number)
{
    int threadsPerBlock = 1024;
    int blocksPerGrid = (number + threadsPerBlock - 1) / threadsPerBlock;
    relu_kernel_cuda<<<blocksPerGrid, threadsPerBlock>>>(input_blob, output_blob, number);
    hipDeviceSynchronize();
}
